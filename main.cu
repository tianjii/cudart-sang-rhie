#include "hip/hip_runtime.h"
/* 
Skylar Sang & Matthew Rhie
ECSE 4740
Spring 2020

Guidance for CUDA, hiprand, and error checking by Nvidia Developer Roger Allen
Project adapted from the book Ray Tracing in One Weekend by Peter Shirley
*/

#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

struct ints3 {
 
    int x;
    int y;
    int z;
   
};


void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


// Function for calculating collisions and color for each pixel
__device__ vec3 write_color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }



 }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int pixel_index = blockIdx.x*blockDim.x + threadIdx.x;
    if (pixel_index >= max_x*max_y) return;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(19+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(ints3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
    int pixel_index = blockIdx.x*blockDim.x + threadIdx.x;
    if (pixel_index >= max_x*max_y) return;
    int j = pixel_index / max_x;
    int i = pixel_index - j*max_x;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += write_color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    ints3 color_int;
    color_int.x = int(255.99*col.x());
    color_int.y = int(255.99*col.y());
    color_int.z = int(255.99*col.z());
    fb[pixel_index] = color_int;
}

#define RND (hiprand_uniform(&local_rand_state))

// Kernel for initializing world

__global__ void random_scene(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000,
                               new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for(int a = -11; a < 11; a++) {
            for(int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hitable_list(d_list, 22*22+1+3);

        vec3 lookfrom(13,2,3);
        vec3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    for(int i=0; i < 22*22+1+3; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

int main(int argc, char** argv) {
    int width = 1200;
    int height = 800;
    int samples = 10;


    int num_pixels = width*height;

    size_t threadsPerBlock = atoi(argv[1]);
    size_t blocks = ceil(num_pixels/threadsPerBlock);
    // allocate FB


    std::cerr << "Rendering a " << width << "x" << height << " image with " << samples << " samples per pixel ";
    std::cerr << "in " << blocks << "blocks with " << threadsPerBlock << " threads.\n";

    ints3 *frame;
    checkCudaErrors(hipMallocManaged((void **)&frame, num_pixels*sizeof(ints3)));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // Initialize random number generator for material/size randomness of the spheres
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Initialize world with 3D hitable objects
    hitable **d_list;
    int num_hitables = 22*22+1+3;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    random_scene<<<1,1>>>(d_list, d_world, d_camera, width, height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Initialize Random Number Generators for each pixel/thread
    render_init<<<blocks, threadsPerBlock>>>(width, height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Run ray tracing algorihtm 
    render<<<blocks, threadsPerBlock>>>(frame, width, height,  samples, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output frame to std output > ppm file
    std::cout << "P3\n" << width << " " << height << "\n255\n";
    for (int i = num_pixels-1; i >= 0; i--) {
            std::cout << frame[i].x << " " << frame[i].y << " " << frame[i].z << "\n";
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(frame));

    hipDeviceReset();
}
